#include "hip/hip_runtime.h"
﻿#include<hip/hip_runtime.h>
#include<>

#include <iostream>
#include"BmpUtile.h"


#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(EXIT_FAILURE); \
    } \
} while (0)

#define CUDA_KERNEL_CHECK() CUDA_CHECK(hipGetLastError())


// Typedef ================

using uchar = unsigned char;


// Consts ================

constexpr int RATIO = 1;
constexpr int BLOCK = 16;


__global__ void BilinearReduce(uchar* dstBuffer, size_t dstPitch, uchar* srcBuffer, size_t srcPitch, int dstWidth, int dstHeight, int srcWidth, int srcHeight)
{

    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;


    if (x < dstWidth && y < dstHeight)
    {
        int srcX = x * (srcWidth / dstWidth);
        int srcY = y * (srcHeight / dstHeight);

        int x1 = srcX;
        int x2 = min(srcX + RATIO, srcWidth - RATIO);
        int y1 = srcY;
        int y2 = min(srcY + RATIO, srcHeight - RATIO);

        uchar c1 = srcBuffer[x1 + srcPitch * y1];
        uchar c2 = srcBuffer[x1 + srcPitch * y2];
        uchar c3 = srcBuffer[x2 + srcPitch * y1];
        uchar c4 = srcBuffer[x2 + srcPitch * y2];
        float value = (c1 + c2 + c3 + c4) * 0.25;

        dstBuffer[x + dstPitch * y] = static_cast<uchar>(value);
    }
}

__global__ void BilinearIncrease(uchar* dstBuffer, size_t dstPitch, uchar* srcBuffer, size_t srcPitch, int dstWidth, int dstHeight, int srcWidth, int srcHeight)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < dstWidth && y < dstHeight)
    {
        int srcX = static_cast<int>(x * (static_cast<float>(srcWidth) / dstWidth));
        int srcY = static_cast<int>(y * (static_cast<float>(srcHeight) / dstHeight));

        int x1 = srcX;
        int x2 = min(srcX + RATIO, srcWidth - RATIO);
        int y1 = srcY;
        int y2 = min(srcY + RATIO, srcHeight - RATIO);

        uchar c1 = srcBuffer[x1 + srcPitch * y1];
        uchar c2 = srcBuffer[x1 + srcPitch * y2];
        uchar c3 = srcBuffer[x2 + srcPitch * y1];
        uchar c4 = srcBuffer[x2 + srcPitch * y2];
        float value = (c1 + c2 + c3 + c4) * 0.25;

        dstBuffer[x + dstPitch * y] = value;
    }
}


int main(void)
{

    // Set Host data =========================================================================================

    uchar* h_rb = nullptr, * h_rbCal = nullptr, * h_rrbb = nullptr;
    uchar* h_gb = nullptr, * h_gbCal = nullptr, * h_ggbb = nullptr;
    uchar* h_bb = nullptr, * h_bbCal = nullptr, * h_bbbb = nullptr;
    int h_width, h_height;

    const char* path = "C:\\Users\\james\\Documents\\2025\\source_code\\lenna.bmp";
    if (!Bmp::BmpToRgbBuffers(path, &h_rb, &h_gb, &h_bb, h_width, h_height))
    {
        if (!h_rb) free(h_rb);
        if (!h_gb) free(h_gb);
        if (!h_bb) free(h_bb);
    }

    h_rbCal = (uchar*)malloc(sizeof(uchar) * (h_width / 2) * (h_height / 2));
    h_gbCal = (uchar*)malloc(sizeof(uchar) * (h_width / 2) * (h_height / 2));
    h_bbCal = (uchar*)malloc(sizeof(uchar) * (h_width / 2) * (h_height / 2));

    h_rrbb = (uchar*)malloc(sizeof(uchar) * h_width * h_height);
    h_ggbb = (uchar*)malloc(sizeof(uchar) * h_width * h_height);
    h_bbbb = (uchar*)malloc(sizeof(uchar) * h_width * h_height);

    // Set Device data ========================================================================================

    int d_width = h_width;
    int d_height = h_height;
    size_t firstPitch, secondPitch;
    uchar* d_rb_first = nullptr, * d_rb_second = nullptr;
    uchar* d_gb_first = nullptr, * d_gb_second = nullptr;
    uchar* d_bb_first = nullptr, * d_bb_second = nullptr;

    CUDA_CHECK(hipMallocPitch(&d_rb_first, &firstPitch, sizeof(uchar) * d_width, d_height));
    CUDA_CHECK(hipMallocPitch(&d_gb_first, &firstPitch, sizeof(uchar) * d_width, d_height));
    CUDA_CHECK(hipMallocPitch(&d_bb_first, &firstPitch, sizeof(uchar) * d_width, d_height));

    CUDA_CHECK(hipMallocPitch(&d_rb_second, &secondPitch, sizeof(uchar) * d_width / 2, d_height / 2));
    CUDA_CHECK(hipMallocPitch(&d_gb_second, &secondPitch, sizeof(uchar) * d_width / 2, d_height / 2));
    CUDA_CHECK(hipMallocPitch(&d_bb_second, &secondPitch, sizeof(uchar) * d_width / 2, d_height / 2));

    CUDA_CHECK(hipMemcpy2D(d_rb_first, firstPitch, h_rb, sizeof(uchar) * h_width, sizeof(char) * h_width, h_height, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy2D(d_gb_first, firstPitch, h_gb, sizeof(uchar) * h_width, sizeof(char) * h_width, h_height, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy2D(d_bb_first, firstPitch, h_bb, sizeof(uchar) * h_width, sizeof(char) * h_width, h_height, hipMemcpyHostToDevice));


    // Run Bilinear Resize =====================================================================================

    dim3 block(BLOCK, BLOCK);
    dim3 grid(((d_width / 2) + BLOCK - 1) / BLOCK, ((d_height / 2) + BLOCK - 1) / BLOCK);


    BilinearReduce <<<grid, block >>> (d_rb_second, secondPitch, d_rb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
    CUDA_KERNEL_CHECK();
    BilinearReduce <<<grid, block >>> (d_gb_second, secondPitch, d_gb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
    CUDA_KERNEL_CHECK();
    BilinearReduce <<<grid, block >>> (d_bb_second, secondPitch, d_bb_first, firstPitch, d_width / 2, d_height / 2, d_width, d_height);
    CUDA_KERNEL_CHECK();

    dim3 grid2 = ((d_width + BLOCK - 1) / BLOCK, (d_height + BLOCK - 1) / BLOCK);

    BilinearIncrease << <grid2, block >> > (d_rb_first, firstPitch, d_rb_second, secondPitch, d_width, d_height, d_width / 2, d_height / 2);
    CUDA_KERNEL_CHECK();
    BilinearIncrease << <grid2, block >> > (d_gb_first, firstPitch, d_gb_second, secondPitch, d_width, d_height, d_width / 2, d_height / 2);
    CUDA_KERNEL_CHECK();
    BilinearIncrease << <grid2, block >> > (d_bb_first, firstPitch, d_bb_second, secondPitch, d_width, d_height, d_width / 2, d_height / 2);
    CUDA_KERNEL_CHECK();

    // Store resized image ======================================================================================

    CUDA_CHECK(hipMemcpy2D(h_rbCal, sizeof(uchar) * (h_width / 2), d_rb_second, secondPitch, d_width / 2, d_height / 2, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_gbCal, sizeof(uchar) * (h_width / 2), d_gb_second, secondPitch, d_width / 2, d_height / 2, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_bbCal, sizeof(uchar) * (h_width / 2), d_bb_second, secondPitch, d_width / 2, d_height / 2, hipMemcpyDeviceToHost));

    const char* outPath = "C:\\Users\\james\\Documents\\2025\\source_code\\reduceLenna.bmp";
    if (!Bmp::RgbBuffersToBmp(outPath, h_rbCal, h_gbCal, h_bbCal, h_width / 2, h_height / 2))
    {
        std::cout << "Error : Writing bmp file failed";
    }

    CUDA_CHECK(hipMemcpy2D(h_rrbb, sizeof(uchar) * h_width, d_rb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_ggbb, sizeof(uchar) * h_width, d_gb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D(h_bbbb, sizeof(uchar) * h_width, d_bb_first, firstPitch, d_width, d_height, hipMemcpyDeviceToHost));

    const char* outPath2 = "C:\\Users\\james\\Documents\\2025\\source_code\\increasedLenna.bmp";
    if (!Bmp::RgbBuffersToBmp(outPath2, h_rrbb, h_ggbb, h_bbbb, h_width, h_height))
    {
        std::cout << "Error : Writing bmp file failed";
    }

    // free =====================================================================================================

    free(h_rb);
    free(h_gb);
    free(h_bb);
    free(h_rbCal);
    free(h_gbCal);
    free(h_bbCal);
    free(h_rrbb);
    free(h_ggbb);
    free(h_bbbb);
    hipFree(d_rb_first);
    hipFree(d_gb_first);
    hipFree(d_bb_first);
    hipFree(d_rb_second);
    hipFree(d_gb_second);
    hipFree(d_bb_second);

    return 0;
}

